#include "hip/hip_runtime.h"
/*
Designer: Carlos Rodriguez Calvo
Date: 16.03.2019
----------------------------------------------------------------------------------------------------------------------
Description:
	-Multipararell GPU library for factorial numbering calculation. 

Code based on: NVIDIA Gforce 1060

-----------------------------------------------------------------------------------------------------------------------
*/

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <conio.h>
#include<time.h>

using namespace std;

__global__ void Factorial(unsigned long long *gpu_n, unsigned long long *gpu_r)
{
	int i;
	*gpu_r = 1;
	for (i = 1; i <= *gpu_n; i++)
	{
		*gpu_r = *gpu_r * i;
	}
}

int main()
{
	hipEvent_t start, stop;
	float totalTime;
	unsigned long long Number;  //to store number on the cpu/host machine
	unsigned long long *dev_number;
	unsigned long long *res, result; //store result 
	printf("\n\t Enter the number : ");
	scanf("%d", &Number);
	//
	hipEventCreate(&start);
	hipEventRecord(start, 0);
	//
	hipMalloc((void**)&dev_number, sizeof(int));
	hipMalloc((void**)&res, sizeof(long int));
	//
	hipMemcpy(dev_number, &Number, sizeof(int), hipMemcpyHostToDevice);
	//
	Factorial << <1, 1 >> > (dev_number, res);
	hipMemcpy(&result, res, sizeof(long int), hipMemcpyDeviceToHost);
	hipEventCreate(&stop);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&totalTime, start, stop);
	printf("Total time : %f ms\n", totalTime);
	printf("\n\t Factorial of number %d is %ld \n", Number, result);
	return 0;
}